#include "cuda_props.h"
#include "cuda_util.h"

hipDeviceProp_t get_deviceProps() {
    static hipDeviceProp_t props;
    static bool gotten = false;

    if (!gotten) CUDA_CHKERR(hipGetDeviceProperties(&props, 0));

    return props;
}

// For use on host
int get_warpSize() {
    return get_deviceProps().warpSize;
}

int get_maxThreadsPerBlock() {
    return get_deviceProps().maxThreadsPerBlock;
}
