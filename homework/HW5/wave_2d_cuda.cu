#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include "png_util.h"
#include "CudaMem.h"
#include "cuda_props.h"
#include "cuda_kernel_interface.h"
#include "bench.h"
#include "debug.h"

//#define min(X,Y) ((X) < (Y) ? (X) : (Y))
//#define max(X,Y) ((X) > (Y) ? (X) : (Y))
#define IDX2D(a, i, stride, j) ((a)[(i)*(stride) + (j)])
#define CHKERR(test, message) \
    IF_DEBUG(if ((test)) fprintf(stderr, "%s:%d: ERROR: %s\n", __FILE__, __LINE__, (message));)

int main(int argc, char ** argv) {
    assert(WARP_SIZE == get_warpSize());

    INIT_CLOCK(setup); INIT_CLOCK(simulation); INIT_CLOCK(file_io); INIT_CLOCK(total);

    START_CLOCK(total); START_CLOCK(setup);
    const int nx = 500;
    const int ny = 500;
    const int mesh_size = nx*ny;
    const int nt = 10000; 
    //int nt = 1000000;
    int frame = 0;
    // fps = 1/(dt*frame_skip)
    int frame_skip = 100;
    int r,c,it;
    double dx,dy,dt;
    double xy_max, xy_min;
    double tmax;
    double dx2inv, dy2inv;
    char filename[sizeof "./images/file00000.png"];

    image_size_t sz; 
    sz.width=nx;
    sz.height=ny;

    CudaMem<double> z(mesh_size);
    CudaMem<double> v(mesh_size);
//    CudaMem<unsigned char> output(mesh_size);
    unsigned char *output = new unsigned char[mesh_size];

//    CudaMem<double> min_scratch(CUDA_MAX_MAX_BLOCKS); CUDA_CHKERR(min_scratch.init_device());
//    CudaMem<double> max_scratch(CUDA_MAX_MAX_BLOCKS); CUDA_CHKERR(max_scratch.init_device());

    xy_max=10.0;
    xy_min=0.0;
    dx = (xy_max-xy_min)/(double)(nx-1);
    dy = (xy_max-xy_min)/(double)(ny-1);
    
    tmax=20.0;
    dt = (tmax-0.0)/(double)(nt-1);

    double x,y; 
    for (r=0;r<ny;r++) {
        for (c=0;c<nx;c++) {
            x = xy_min+(double)c*dx;
            y = xy_min+(double)r*dy;
            IDX2D(z.host_ptr(), r, nx, c) = exp(-(sqrt((x-5.0)*(x-5.0)+(y-5.0)*(y-5.0))));
            IDX2D(v.host_ptr(), r, nx, c) = 0.0;
        }
    }

    CudaKernelParams sim_params = sim_kernel_params(nx, ny);
//    CudaKernelParams min_max_params = min_max_kernel_params(nx, ny);

    CUDA_CHKERR(z.to_device()); CUDA_CHKERR(v.to_device());

    STOP_CLOCK(setup);

    printf("nt=%d, dt=%g, frame_skip=%d, fps=%g\n", nt, dt, frame_skip, 1/(dt*frame_skip));

    START_CLOCK(simulation);
    dx2inv = 1.0/(dx*dx);
    dy2inv = 1.0/(dy*dy);

    for(it=0;it<nt-1;it++) {
        launch_sim_kernel_tiled(z.device_ptr(), v.device_ptr(), nx, ny, dx2inv, dy2inv, dt,
                                sim_params);
        IF_DEBUG(CUDA_CHKERR(hipGetLastError());)
        IF_DEBUG(CUDA_CHKERR(hipDeviceSynchronize());)
//        for (r=1;r<ny-1;r++)  
//            for (c=1;c<nx-1;c++) {
//                const double z_val =    IDX2D(z, r,   nx, c);
//                const double z_x_high = IDX2D(z, r+1, nx, c);
//                const double z_x_low =  IDX2D(z, r-1, nx, c);
//                const double z_y_high = IDX2D(z, r,   nx, c+1);
//                const double z_y_low =  IDX2D(z, r,   nx, c-1);
//                const double ax = (z_x_high+z_x_high-2.0*z_val)*dx2inv;
//                const double ay = (z_y_high+z_y_low-2.0*z_val)*dy2inv;
//                IDX2D(a, r, nx, c) = (ax+ay)/2;
//            }
//        for (r=1; r<ny-1; r++)  
//            for (c=1;c<nx-1;c++) {
//                IDX2D(v, r, nx, c) = IDX2D(v, r, nx, c) + dt*IDX2D(a, r, nx, c);
//                IDX2D(z, r, nx, c) = IDX2D(z, r, nx, c) + dt*IDX2D(v, r, nx, c);
//            }

        if (it % frame_skip == 0) {
            CUDA_CHKERR(z.to_host());

            double mx,mn;
            mx = -999999;
            mn = 999999;
            for (size_t k = 0; k < mesh_size; ++k) {
                mx = max(mx, z[k]);
                mn = min(mn, z[k]);
            }

            for (size_t k=0; k < mesh_size; ++k)
                output[k] = (char) round((z[k]-mn)/(mx-mn)*255);

//            cuda_min_max(z.device_ptr(), z.size(), min_scratch.device_ptr(), max_scratch.device_ptr());
//            min_scratch.to_host(0, 1); max_scratch.to_host(0, 1);
//            const double mn = min_scratch[0];
//            const double mx = max_scratch[0];
//            cuda_grayscale(z.device_ptr(), output.device_ptr(), mesh_size, mn, mx);

            STOP_CLOCK(simulation);
            START_CLOCK(file_io);

            sprintf(filename, "./images/file%05d.png", frame);
            printf("Writing %s\n",filename);    
            write_png_file(filename,output/*.host_ptr()*/,sz);

            STOP_CLOCK(file_io);
            START_CLOCK(simulation);

            frame+=1;
        }
    }
    
    CUDA_CHKERR(z.to_host());

    double mx,mn;
    mx = -999999;
    mn = 999999;
    for (size_t k = 0; k < mesh_size; ++k) {
        mx = max(mx, z[k]);
        mn = min(mn, z[k]);
    }

    for (size_t k = 0; k < mesh_size; ++k)
        output[k] = (char) round((z[k]-mn)/(mx-mn)*255);

//    cuda_min_max(z.device_ptr(), z.size(), min_scratch.device_ptr(), max_scratch.device_ptr());
//    min_scratch.to_host(0, 1); max_scratch.to_host(0, 1);
//    const double mn = min_scratch[0];
//    const double mx = max_scratch[0];
//    cuda_grayscale(z.device_ptr(), output.device_ptr(), mesh_size, mn, mx);

    printf("%f, %f\n", mn, mx);

    STOP_CLOCK(simulation);

    START_CLOCK(file_io);
    sprintf(filename, "./images/file%05d.png", it);
    printf("Writing %s\n",filename);    
    //Write out output image using 1D serial pointer
    write_png_file(filename,output/*.host_ptr()*/,sz);
    STOP_CLOCK(file_io); STOP_CLOCK(total);

#ifdef BENCH
    fputs("BENCHMARKING\nTOTAL setup file_io simulation\n", stderr);
    fprintf(stderr, "%.3e %.3e %.3e %.3e\n", total_time, setup_time, file_io_time, simulation_time);
#endif

    return 0;
}
